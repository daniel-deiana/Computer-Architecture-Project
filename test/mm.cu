#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void mm(int* c, int* a, int* b, int n){

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x > n || y > n)
    return;

  for (int k = 0; k < n; k++)
    c[x * n + y] += a[x * n + k] * b[k * n + y];
}

int main (void){

  int DIM_V = 32;

  int a[DIM_V][DIM_V],b[DIM_V][DIM_V],c[DIM_V][DIM_V];
  int*d_a, *d_b, *d_c;

  dim3 block(DIM_V, DIM_V);

  int size = sizeof(int)*DIM_V*DIM_V;

  // allocate memory on device
  hipMalloc((void**)&d_a,size);
  hipMalloc((void**)&d_b,size);
  hipMalloc((void**)&d_c,size);

  for (int k = 0; k < DIM_V; k++) {
    for (int j = 0; j < DIM_V; j++) {
    a[k][j] = 1;
    b[k][j] = 1;
    c[k][j] = 0;
    }
  }

  hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
  hipMemcpy(d_c,&c,size,hipMemcpyHostToDevice);

  mm<<<1,block>>> (d_c,d_a,d_b,DIM_V);

  hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);


   for (int k = 0; k < DIM_V; k++) {
    for (int j = 0; j < DIM_V; j++) {
    printf("%d ",c[k][j]);
    }
    printf("\n");
  }
  hipFree(d_a);hipFree(d_b);hipFree(d_c);

return 0;
}